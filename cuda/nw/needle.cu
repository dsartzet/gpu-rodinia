#define LIMIT -999
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "needle.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>

// includes, kernels
#include "needle_kernel.cu"

#ifdef TIMING
#include "timing.h"

struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);


int blosum62[24][24] = {
{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
};

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

  printf("WG size of kernel = %d \n", BLOCK_SIZE);

    runTest( argc, argv);

    return EXIT_SUCCESS;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <max_rows/max_cols> <penalty> \n", argv[0]);
	fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
	fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
	exit(1);
}

void runTest( int argc, char** argv) 
{
  const char *results = "print traceback value GPU:\n\
2 12 6 8 18 12 15 7 7 2 -7 -8 2 -7 -9 -9 1 -5 -13 -13 -22 -12 -18 -15 -12 -20 -20 -18 -20 -16 -14 -11 -8 -13 -12 -20 -20 -16 -16 -12 -8 -4 -6 -4 -13 -13 -13 -3 -8 -14 -23 -27 -26 -26 -23 -19 -18 -27 -17 -25 -24 -33 -37 -27 -17 -23 -21 -23 -27 -24 -21 -23 -20 -20 -18 -14 -19 -16 -13 -12 -9 -15 -13 -22 -22 -22 -22 -19 -17 -17 -13 -13 -21 -20 -20 -20 -25 -25 -22 -19 -21 -25 -34 -43 -40 -42 -40 -36 -44 -42 -43 -43 -43 -40 -38 -37 -34 -39 -36 -37 -43 -43 -48 -48 -47 -45 -50 -49 -46 -43 -41 -38 -35 -33 -41 -40 -30 -28 -34 -31 -31 -40 -37 -33 -39 -38 -37 -35 -36 -42 -42 -42 -40 -37 -37 -37 -34 -31 -33 -29 -38 -43 -43 -48 -50 -47 -46 -46 -46 -50 -48 -46 -51 -51 -49 -47 -51 -51 -48 -50 -50 -46 -48 -44 -46 -45 -54 -51 -47 -49 -48 -45 -44 -45 -45 -41 -38 -43 -43 -42 -41 -31 -32 -29 -26 -26 -31 -28 -24 -23 -20 -26 -24 -29 -19 -28 -30 -30 -28 -27 -26 -26 -25 -33 -30 -39 -29 -31 -22 -22 -21 -21 -17 -18 -15 -12 -16 -13 -17 -17 -15 -11 -11 -11 -16 -13 -11 -11 -20 -26 -28 -28 -26 -26 -30 -30 -34 -34 -34 -31 -21 -23 -31 -31 -30 -28 -28 -26 -25 -25 -33 -34 -34 -39 -35 -33 -37 -34 -35 -33 -30 -29 -33 -23 -28 -24 -14 -14 -19 -9 -7 -15 -11 -8 -13 -19 -9 1 0 ";

  int max_rows, max_cols, penalty;
  int *input_itemsets, *output_itemsets, *referrence;
	int *matrix_cuda,  *referrence_cuda;
	int size;
    
    // the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 3)
	{
		max_rows = atoi(argv[1]);
		max_cols = atoi(argv[1]);
		penalty = atoi(argv[2]);
	}
    else{
	usage(argc, argv);
    }
	
	if(atoi(argv[1])%16!=0){
	fprintf(stderr,"The dimension values must be a multiple of 16\n");
	exit(1);
	}
	

	max_rows = max_rows + 1;
	max_cols = max_cols + 1;
	referrence = (int *)malloc( max_rows * max_cols * sizeof(int) );
    input_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );
	output_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );
	

	if (!input_itemsets)
		fprintf(stderr, "error: can not allocate memory");

    srand ( 7 );
	
	
    for (int i = 0 ; i < max_cols; i++){
		for (int j = 0 ; j < max_rows; j++){
			input_itemsets[i*max_cols+j] = 0;
		}
	}
	
	printf("Start Needleman-Wunsch\n");
	
	for( int i=1; i< max_rows ; i++){    //please define your own sequence. 
       input_itemsets[i*max_cols] = rand() % 10 + 1;
	}
    for( int j=1; j< max_cols ; j++){    //please define your own sequence.
       input_itemsets[j] = rand() % 10 + 1;
	}


	for (int i = 1 ; i < max_cols; i++){
		for (int j = 1 ; j < max_rows; j++){
		referrence[i*max_cols+j] = blosum62[input_itemsets[i*max_cols]][input_itemsets[j]];
		}
	}

    for( int i = 1; i< max_rows ; i++)
       input_itemsets[i*max_cols] = -i * penalty;
	for( int j = 1; j< max_cols ; j++)
       input_itemsets[j] = -j * penalty;


    size = max_cols * max_rows;
	hipMalloc((void**)& referrence_cuda, sizeof(int)*size);
	hipMalloc((void**)& matrix_cuda, sizeof(int)*size);
	
	hipMemcpy(referrence_cuda, referrence, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size, hipMemcpyHostToDevice);

    dim3 dimGrid;
	dim3 dimBlock(BLOCK_SIZE, 1);
	int block_width = ( max_cols - 1 )/BLOCK_SIZE;

#ifdef  TIMING
  gettimeofday(&tv_kernel_start, NULL);
#endif

	printf("Processing top-left matrix\n");
	//process top-left matrix
	for( int i = 1 ; i <= block_width ; i++){
		dimGrid.x = i;
		dimGrid.y = 1;
		needle_cuda_shared_1<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda
		                                      ,max_cols, penalty, i, block_width); 
	}
	printf("Processing bottom-right matrix\n");
    //process bottom-right matrix
	for( int i = block_width - 1  ; i >= 1 ; i--){
		dimGrid.x = i;
		dimGrid.y = 1;
		needle_cuda_shared_2<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda
		                                      ,max_cols, penalty, i, block_width); 
	}

#ifdef  TIMING
    gettimeofday(&tv_kernel_end, NULL);
    tvsub(&tv_kernel_end, &tv_kernel_start, &tv);
    kernel_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

    hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size, hipMemcpyDeviceToHost);

#define TRACEBACK
#ifdef TRACEBACK

	// FILE *fpo = fopen("result.txt","w");
	// fprintf(fpo, "print traceback value GPU:\n");
  char *str_result = new char[(sizeof(char) * (strlen(results)+1))];
  str_result[0] = 0;

  sprintf(str_result + strlen(str_result), "print traceback value GPU:\n");

	for (int i = max_rows - 2,  j = max_rows - 2; i>=0, j>=0;){
		int nw, n, w, traceback;
		if ( i == max_rows - 2 && j == max_rows - 2 )
			// fprintf(fpo, "%d ", output_itemsets[ i * max_cols + j]); //print the first element
      sprintf(str_result + strlen(str_result), "%d ", output_itemsets[ i * max_cols + j]);
		if ( i == 0 && j == 0 )
           break;
		if ( i > 0 && j > 0 ){
			nw = output_itemsets[(i - 1) * max_cols + j - 1];
		    w  = output_itemsets[ i * max_cols + j - 1 ];
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else if ( i == 0 ){
		    nw = n = LIMIT;
		    w  = output_itemsets[ i * max_cols + j - 1 ];
		}
		else if ( j == 0 ){
		    nw = w = LIMIT;
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else{
		}

		//traceback = maximum(nw, w, n);
		int new_nw, new_w, new_n;
		new_nw = nw + referrence[i * max_cols + j];
		new_w = w - penalty;
		new_n = n - penalty;
		
		traceback = maximum(new_nw, new_w, new_n);
		if(traceback == new_nw)
			traceback = nw;
		if(traceback == new_w)
			traceback = w;
		if(traceback == new_n)
            traceback = n;
		// fprintf(fpo, "%d ", traceback);
    sprintf(str_result + strlen(str_result), "%d ", traceback);

		if(traceback == nw )
		{i--; j--; continue;}

        else if(traceback == w )
		{j--; continue;}

        else if(traceback == n )
		{i--; continue;}

		else
		;
	}

  if (strcmp(results, str_result) == 0) {
      printf("Test PASSED\n");
    } else {
      printf("Test FAILED\n");
    }

	// fclose(fpo);
  delete str_result;

#endif

	hipFree(referrence_cuda);
	hipFree(matrix_cuda);

	free(referrence);
	free(input_itemsets);
	free(output_itemsets);

#ifdef  TIMING
    printf("Exec: %f\n", kernel_time);
#endif
}
