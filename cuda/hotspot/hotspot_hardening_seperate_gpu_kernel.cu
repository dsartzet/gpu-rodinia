
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#ifdef RD_WG_SIZE_0_0                                                            
        #define BLOCK_SIZE RD_WG_SIZE_0_0                                        
#elif defined(RD_WG_SIZE_0)                                                      
        #define BLOCK_SIZE RD_WG_SIZE_0                                          
#elif defined(RD_WG_SIZE)                                                        
        #define BLOCK_SIZE RD_WG_SIZE                                            
#else                                                                                    
        #define BLOCK_SIZE 16                                                            
#endif                                                                                   

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)



void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

void writeoutput(float *vect, int grid_rows, int grid_cols, char *file){

	int i,j, index=0;
	FILE *fp;
	char str[STR_SIZE];

	if( (fp = fopen(file, "w" )) == 0 )
          printf( "The file was not opened\n" );


	for (i=0; i < grid_rows; i++) 
	 for (j=0; j < grid_cols; j++)
	 {

		 sprintf(str, "%d\t%g\n", index, vect[i*grid_cols+j]);
		 fputs(str,fp);
		 index++;
	 }
		
      fclose(fp);	
}


void readinput(float *vect, int grid_rows, int grid_cols, char *file){

  	int i,j;
	FILE *fp;
	char str[STR_SIZE];
	float val;

	if( (fp  = fopen(file, "r" )) ==0 )
            printf( "The file was not opened\n" );


	for (i=0; i <= grid_rows-1; i++) 
	 for (j=0; j <= grid_cols-1; j++)
	 {
		fgets(str, STR_SIZE, fp);
		if (feof(fp))
			fatal("not enough lines in file");
		//if ((sscanf(str, "%d%f", &index, &val) != 2) || (index != ((i-1)*(grid_cols-2)+j-1)))
		if ((sscanf(str, "%f", &val) != 1))
			fatal("invalid file format");
		vect[i*grid_cols+j] = val;
	}

	fclose(fp);	

}


/* START of Lishan add */
__global__ void check_correctness(float* result)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
    if (result[tid] != result[tid+65536]) // if copy0 and copy1 are different
    {
    	// two cases:
		if (result[tid] != result[tid+65536*2] && result[tid+65536]!= result[tid+65536*2])
		{ 
			// 1. not correctable.
		    // All three copies have different results. This is considered as DUE, not SDC.
		    printf ("DUE %f %f %f\n", result[tid], result[tid+65536], result[tid+65536*2]);  
		}
		else
		{
			// 2. correctable, majority voting
		    // printf ("correcting tid=%d %.10f %.10f %.10f\n", tid,result[tid], result[tid+65536], result[tid+65536*2]);  
		    result[tid] = result[tid+65536*2];
		}
    }   
}
/* END of Lishan add */

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(int iteration,  //number of iteration
                               float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               int grid_cols,  //Col of grid
                               int grid_rows,  //Row of grid
							   int border_cols,  // border offset 
							   int border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx, 
                               float Ry, 
                               float Rz, 
                               float step, 
                               float time_elapsed){
	
	/* START of Lishan modify */
	// we have 3x threads on tx, so this shared_mem[ty][tx], tx dimension should x3.
        __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE*3];
        __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE*3];
        __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE*3]; // saving temparary temperature result
	/* END of Lishan modify */

	/* START of original code 
	__shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result
	END of original code */

	float amb_temp = 80.0;
        float step_div_Cap;
        float Rx_1,Ry_1,Rz_1;
        
	int bx = blockIdx.x; 
        int by = blockIdx.y;

	// From this point, tx is not thread id, but a reference to locate the data
	int tx=threadIdx.x % BLOCK_SIZE; // modified by Lishan. 
//	int tx=threadIdx.x; // original code
	int ty=threadIdx.y;
	
	int cur_copy = threadIdx.x / BLOCK_SIZE; // added by Lishan.
	step_div_Cap=step/Cap;
	
	Rx_1=1/Rx;
	Ry_1=1/Ry;
	Rz_1=1/Rz;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_rows = BLOCK_SIZE-iteration*2;//EXPAND_RATE
	int small_block_cols = BLOCK_SIZE-iteration*2;//EXPAND_RATE

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkY = small_block_rows*by-border_rows;
        int blkX = small_block_cols*bx-border_cols;
        int blkYmax = blkY+BLOCK_SIZE-1;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int yidx = blkY+ty;
	int xidx = blkX+tx;

        // load data if it is within the valid input range
	int loadYidx=yidx, loadXidx=xidx;
        int index = grid_cols*loadYidx+loadXidx;
       
	if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
            temp_on_cuda[ty][threadIdx.x] = temp_src[index];  // Load the temperature data from global memory to shared memory // modified by Lishan
            power_on_cuda[ty][threadIdx.x] = power[index];// Load the power data from global memory to shared memory // modified by Lishan. These things should be loaded 3 times.
	//    temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory // original code
        //    power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory // original code
	}
	__syncthreads();

        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validYmin = (blkY < 0) ? -blkY : 0;
        int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

        int N = ty-1;
        int S = ty+1;
        int W = tx-1;
        int E = tx+1;
        
        N = (N < validYmin) ? validYmin : N;
        S = (S > validYmax) ? validYmax : S;
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(tx, validXmin, validXmax) && \
                  IN_RANGE(ty, validYmin, validYmax) ) {
                  computed = true;

		/* START of Lishan modify */
		// each thread should use the data loaded and computed by themself.
                  temp_t[ty][threadIdx.x] =   temp_on_cuda[ty][threadIdx.x] + step_div_Cap * (power_on_cuda[ty][threadIdx.x] + 
	       	         (temp_on_cuda[S][threadIdx.x] + temp_on_cuda[N][threadIdx.x] - 2.0*temp_on_cuda[ty][threadIdx.x]) * Ry_1 + 
		             (temp_on_cuda[ty][E+cur_copy*BLOCK_SIZE] + temp_on_cuda[ty][W+cur_copy*BLOCK_SIZE] - 2.0*temp_on_cuda[ty][threadIdx.x]) * Rx_1 + 
		             (amb_temp - temp_on_cuda[ty][threadIdx.x]) * Rz_1);


	/* ***** debug *
if (bx == 10 && (by == 20) &&(threadIdx.x==16) && (threadIdx.y == 4)){ 
printf ("LSD copy 1 %.10f %.10f %.10f %.10f %.10f %.10f %.10f \n", temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+0*BLOCK_SIZE] , temp_on_cuda[ty][W+0*BLOCK_SIZE]);
printf ("LSD copy 2 %.10f %.10f %.10f %.10f %.10f %.10f %.10f \n", temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+1*BLOCK_SIZE] , temp_on_cuda[ty][W+1*BLOCK_SIZE]);
printf ("LSD copy 3 %.10f %.10f %.10f %.10f %.10f %.10f %.10f \n", temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+2*BLOCK_SIZE] , temp_on_cuda[ty][W+2*BLOCK_SIZE]);

}

       * ***** debug */



		/* END of Lishan modify */

		 /* START of original code 
	         temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
	       	         (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 + 
		             (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 + 
		             (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
		 END of original code */

	
            }

/* ***** debug */ 
if (bx == 10 && (by == 20)){// &&(threadIdx.x==16) && (threadIdx.y == 4)){ 
printf ("2LSD copy 1 b=(%d, %d), t=(%d,%d)\t%.10f %.10f %.10f %.10f %.10f %.10f %.10f curi=%d\n", bx, by, threadIdx.x, threadIdx.y, temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+0*BLOCK_SIZE] , temp_on_cuda[ty][W+0*BLOCK_SIZE], i);
printf ("2LSD copy 2 b=(%d, %d), t=(%d,%d)\t%.10f %.10f %.10f %.10f %.10f %.10f %.10f \n", bx, by, threadIdx.x, threadIdx.y,temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+1*BLOCK_SIZE] , temp_on_cuda[ty][W+1*BLOCK_SIZE]);
printf ("2LSD copy 3 b=(%d, %d), t=(%d,%d)\t%.10f %.10f %.10f %.10f %.10f %.10f %.10f \n", bx, by, threadIdx.x, threadIdx.y,temp_t[ty][threadIdx.x], temp_on_cuda[ty][threadIdx.x], power_on_cuda[ty][threadIdx.x] ,temp_on_cuda[S][threadIdx.x], temp_on_cuda[N][threadIdx.x], temp_on_cuda[ty][E+2*BLOCK_SIZE] , temp_on_cuda[ty][W+2*BLOCK_SIZE]);

}

/* ***** debug */


            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
        	temp_on_cuda[ty][threadIdx.x]= temp_t[ty][threadIdx.x]; // modified by Lishan
                // temp_on_cuda[ty][tx]= temp_t[ty][tx]; // original code
            __syncthreads();
          }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
 	    /* START of Lishan modify, checking code in the same kernel.
		if (threadIdx.x / BLOCK_SIZE == 0) // master thread
		{
		// only the master thread compare results and determine the correct answer
		// for master thread, tx = threadIdx.x
			if (temp_t[ty][tx] == temp_t[ty][tx+BLOCK_SIZE]) // if copy 0 == copy 1
				temp_dst[index] = temp_t[ty][tx];
			else    // copy 0 != copy 1, then copy 2 is correct
				temp_dst[index] = temp_t[ty][tx+BLOCK_SIZE*2];
		}
	     END of Lishan modify */
        temp_dst[index+cur_copy*65536]= temp_t[ty][threadIdx.x];	// modified by Lishan.
        // temp_dst[index]= temp_t[ty][tx];	// original code
      }
}

/*
   compute N time steps
*/

int compute_tran_temp(float *MatrixPower,float *MatrixTemp[2], int col, int row, \
		int total_iterations, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows, int size, float *MatrixOut) 
{
        dim3 dimBlock(BLOCK_SIZE*3, BLOCK_SIZE); // modified by Lishan: threads on x-dim, x3
//        dim3 dimBlock(BLOCK_SIZE*3, BLOCK_SIZE); // original code
        dim3 dimGrid(blockCols, blockRows);  

	dim3 dimBlockHardening(256); // added by Lishan
	dim3 dimGridHardening(65536/256); // added by Lishan


	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;
	float t;
        float time_elapsed;
	time_elapsed=0.001;

        int src = 1, dst = 0;
	
	for (t = 0; t < total_iterations; t+=num_iterations) {
            int temp = src;
            src = dst;
            dst = temp;
            calculate_temp<<<dimGrid, dimBlock>>>(MIN(num_iterations, total_iterations-t), MatrixPower,MatrixTemp[src],MatrixTemp[dst],\
		col,row,borderCols, borderRows, Cap,Rx,Ry,Rz,step,time_elapsed);

	    /* START of Lishan adding */
	   
	    check_correctness<<<dimGridHardening, dimBlockHardening>>>(MatrixTemp[dst]);	

	    /* END of Lishan adding */
/* cpu checking. This needs to be checked after every kernel - Next kernel is using the results
   cudaMemcpy(MatrixOut, MatrixTemp[dst], sizeof(float)*size*3, cudaMemcpyDeviceToHost); // L
// aL
//printf ("hiLishan %d %d\n", num_iterations, total_iterations);
    for (int il = 0; il < 65536; il++)
    {
        if (MatrixOut[il] != MatrixOut[il+65536])
        {
	//	printf ("diffffffffffffffffffff il=%d, %f %f %f\n", il, MatrixOut[il], MatrixOut[il+65536], MatrixOut[il+65536*2]);
		MatrixOut[il] = MatrixOut[il+65536*2];

        }
    }
   cudaMemcpy(MatrixTemp[dst], MatrixOut, sizeof(float)*size*3, cudaMemcpyHostToDevice); // L
*/

	}

        return dst;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
	fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
	fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
	fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char** argv)
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    int size;
    int grid_rows,grid_cols;
    float *FilesavingTemp,*FilesavingPower,*MatrixOut; 
    char *tfile, *pfile, *ofile;
    
    int total_iterations = 60;
    int pyramid_height = 1; // number of iterations
	
	if (argc != 7)
		usage(argc, argv);
	if((grid_rows = atoi(argv[1]))<=0||
	   (grid_cols = atoi(argv[1]))<=0||
       (pyramid_height = atoi(argv[2]))<=0||
       (total_iterations = atoi(argv[3]))<=0)
		usage(argc, argv);
		
	tfile=argv[4];
    pfile=argv[5];
    ofile=argv[6];
	
    size=grid_rows*grid_cols;

    /* --------------- pyramid parameters --------------- */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (pyramid_height)*EXPAND_RATE/2;
    int borderRows = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    int blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    FilesavingTemp = (float *) malloc(size*sizeof(float));
    FilesavingPower = (float *) malloc(size*sizeof(float));
    MatrixOut = (float *) calloc (size, sizeof(float)*3); // L

    if( !FilesavingPower || !FilesavingTemp || !MatrixOut)
        fatal("unable to allocate memory");

    printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",\
	pyramid_height, grid_cols, grid_rows, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);
	
    readinput(FilesavingTemp, grid_rows, grid_cols, tfile);
    readinput(FilesavingPower, grid_rows, grid_cols, pfile);

    float *MatrixTemp[2], *MatrixPower;
    hipMalloc((void**)&MatrixTemp[0], sizeof(float)*size);
    hipMalloc((void**)&MatrixTemp[1], sizeof(float)*size*3); // L
    hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);

    hipMalloc((void**)&MatrixPower, sizeof(float)*size);
    hipMemcpy(MatrixPower, FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);
    printf("Start computing the transient temperature\n");
    int ret = compute_tran_temp(MatrixPower,MatrixTemp,grid_cols,grid_rows, \
	 total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows, size, MatrixOut);
	printf("Ending simulation\n");



    hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float)*size, hipMemcpyDeviceToHost);

    writeoutput(MatrixOut,grid_rows, grid_cols, ofile);




    hipFree(MatrixPower);
    hipFree(MatrixTemp[0]);
    hipFree(MatrixTemp[1]);
    free(MatrixOut);
}
