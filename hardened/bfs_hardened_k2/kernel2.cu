#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL2_H_
#define _KERNEL2_H_

__global__ void
Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over, int no_of_nodes)
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes && g_updating_graph_mask[tid+blockIdx.y*32768])
	{

		g_graph_mask[tid+blockIdx.y*32768]=true;
		g_graph_visited[tid+blockIdx.y*32768]=true;
		//*g_over=true;
		g_over[blockIdx.y] = true;
		g_updating_graph_mask[tid+blockIdx.y*32768]=false;
	}
}


/* START of Lishan add */
__global__ void prepare_dup_bool(bool* a, int size)
{
	
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    a[tid+size] = a[tid];
    a[tid+size*2] = a[tid];
}
 
__global__ void check_correctness(bool* result, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    if (result[tid] != result[tid+size])
    {
        if (result[tid] != result[tid+size*2] && result[tid+size]!= result[tid+size*2])
        {
            printf ("DUE %d %d %d\n", result[tid], result[tid+size], result[tid+size*2]);
            // All three copies have different results. This is considered as DUE, not SDC.
        }
        else
        {
          //printf ("bool correcting tid=%d %d %d %d\n", tid,result[tid], result[tid+size], result[tid+size*2]);  
            result[tid] = result[tid+size*2];
        }
    }
}

/* END of Lishan add */




#endif

