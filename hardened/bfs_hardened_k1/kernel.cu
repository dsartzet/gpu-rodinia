#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void
Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes) 
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes && g_graph_mask[tid + blockIdx.y*32768])
	{
		g_graph_mask[tid+ blockIdx.y*32768]=false;
		for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
			{
			int id = g_graph_edges[i];
			if(!g_graph_visited[id])
				{
				g_cost[id + blockIdx.y*32768]=g_cost[tid + blockIdx.y*32768]+1;
				g_updating_graph_mask[id + blockIdx.y*32768]=true;
				}
			}
	}
}

/* START of Lishan add */

__global__ void prepare_dup_int(int* a, int size)
{
	
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    a[tid+size] = a[tid];
    a[tid+size*2] = a[tid];
}
 
__global__ void check_correctness_int(int* result, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    if (result[tid] != result[tid+size])
    {
        if (result[tid] != result[tid+size*2] && result[tid+size]!= result[tid+size*2])
        {
            printf ("DUE %d %d %d\n", result[tid], result[tid+size], result[tid+size*2]);
            // All three copies have different results. This is considered as DUE, not SDC.
        }
        else
        {
//          printf ("correcting tid=%d %d %d %d\n", tid,result[tid], result[tid+size], result[tid+size*2]);  
            result[tid] = result[tid+size*2];
        }
    }
}



__global__ void prepare_dup_bool(bool* a, int size)
{
	
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    a[tid+size] = a[tid];
    a[tid+size*2] = a[tid];
}
 
__global__ void check_correctness(bool* result, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    if (result[tid] != result[tid+size])
    {
        if (result[tid] != result[tid+size*2] && result[tid+size]!= result[tid+size*2])
        {
            printf ("DUE %d %d %d\n", result[tid], result[tid+size], result[tid+size*2]);
            // All three copies have different results. This is considered as DUE, not SDC.
        }
        else
        {
//          printf ("bool correcting tid=%d %d %d %d\n", tid,result[tid], result[tid+size], result[tid+size*2]);  
            result[tid] = result[tid+size*2];
        }
    }
}

/* END of Lishan add */



#endif 
